#include "hip/hip_runtime.h"
#include "GPUHeader.h"
#include "GPUConfig.h"

// global variables-----------------------------
float *Zout_hst, *MNout_hst;
float *MNdat_hst, *Zdat_hst;
float *R24_hst, *R35_hst, *H_hst;
float *R_MASS_hst;
float *Zmax_hst;
// __device__ float *R35_dev;
// __device__ float *R24_dev, *H_dev;
// __device__ float *Z_dat_dev, *MN_dat_dev;
__device__ float *MN_out_dev, *Z_out_dev;
__constant__ __device__ uint32_t size_dev[4];
// texture<float, hipTextureType2D, hipReadModeElementType> ZtexRef;
// texture<float, hipTextureType2D, hipReadModeElementType> MNtexRef;


hipStream_t EXECstream[NUMSTREAM];
dim3 DimBlockMomt_MN(BLOCKX_MOMT,1,1);
dim3 DimGridMomt_MN(0,1,1);
dim3 DimBlockMomt(BLOCKX,BLOCKY,2);
dim3 DimGridMomt(0,0,1);
dim3 DimBlockMass(BLOCKX_MASS,1,1);
dim3 DimGridMass(1,1,1);
dim3 DimBlockOpenBD(BLOCKX_OPENBD,1,1);
dim3 DimGridOpenBD_LR(0,1,1);
dim3 DimGridOpenBD_TB(0,1,1);
size_t GridMaxAmp;

uint32_t size_hst[4];
hipDeviceProp_t dev_prop;
#ifdef DEBUG
    float *tmpout;
#endif
//----------------------------------------------
extern "C" void cuda_boot_(float*,float*,float*,float*,float*,float*,float*,float*,float*,int*,int*);
extern "C" void cuda_update_(void);
void cudaMalloc2E(void**, void*, void*, size_t);
#ifdef DEBUG
    __host__ __device__ void prt_mat(float*, size_t, size_t);
    __global__ void GPUmemtest(void);
    __global__ void CHECK_VAR(void);
    extern "C" void cmpvar_(const float *, int *);
#endif


extern "C" void cuda_boot_(float *R1_f, float *R2_f, float *R3_f, float *R4_f, float *R5_f, \
                           float *R6_f, float *R11_f, float *H_f, float *Z_f, int *row, int *col){

    //float *R24_hst, *R35_hst, *H_hst;
    float *R3, *R5;
    float *R_MASS;



    size_hst[0] = *row;
    size_hst[1] = *col;
    size_hst[2] = (*row)*(*col);
    size_hst[3] = (*row)*(*col)*sizeof(float);
    cudaCHK( hipGetDeviceProperties(&dev_prop, 0) );
    printf("GPU INFORMATIONS:                      %s\n", dev_prop.name);
    printf("-->Compute Capabilities [Major.Miner]: %d.%d\n", dev_prop.major, dev_prop.minor);
    printf("-->Clock Rate:                         %d\n", dev_prop.clockRate);
    printf("-->Streaming Multi-Processor Count:    %d\n", dev_prop.multiProcessorCount);
    printf("-->Shared Memory size per SM           %d\n", dev_prop.sharedMemPerBlock);
    printf("-->Total Constant Memory size:         %d\n", dev_prop.totalConstMem );
    printf("-->Maximum Grid Size:                  %dx%dx%d\n", dev_prop.maxGridSize[0],dev_prop.maxGridSize[1],dev_prop.maxGridSize[2]);
    printf("-->Warp Size:                          %d\n", dev_prop.warpSize);
    // allocate variables
    // const === R2 R4 H ===
    cudaCHK( hipMalloc(&H_hst, size_hst[3]) );

    cudaMalloc2E((void**)&R24_hst, R2_f, R4_f, size_hst[3]);
    //cudaCHK( hipMemcpyToSymbol(HIP_SYMBOL(R24_dev), &R24_hst, sizeof(float*)) );
    // const variables R3 R5, R1 R6 R11
    size_t R_size = sizeof(float)*size_hst[1];
    //R1 = (float*) malloc(R_size);
    R3 = (float*) malloc(R_size);
    R5 = (float*) malloc(R_size);
    //R11 =(float*) malloc(R_size);
    R_MASS = (float*) malloc(4*R_size);

    for (size_t i = 0; i < size_hst[1]; i++) {
        //R1[i] = R1_f[i*size_hst[0]];
        R3[i] = R3_f[i*size_hst[0]];
        R5[i] = R5_f[i*size_hst[0]];
        //R11[i]= R11_f[i*size_hst[0]];
        R_MASS[4*i]   = R1_f[i*size_hst[0]];
        R_MASS[4*i+1] = R6_f[i];
        R_MASS[4*i+2] = R11_f[i*size_hst[0]];
    }


    // cudaCHK( hipMalloc(&R1_hst, R_size) );
    // cudaCHK( hipMemcpy(R1_hst, R1, R_size, hipMemcpyHostToDevice) );

    // cudaCHK( hipMalloc(&R6_hst, R_size) );
    // cudaCHK( hipMemcpy(R6_hst, R6_f, R_size, hipMemcpyHostToDevice) );

    // cudaCHK( hipMalloc(&R11_hst, R_size) );
    // cudaCHK( hipMemcpy(R11_hst, R11, R_size, hipMemcpyHostToDevice) );
    cudaCHK( hipMalloc(&R_MASS_hst, 4*R_size) );
    cudaCHK( hipMemcpy(R_MASS_hst, R_MASS, 4*R_size, hipMemcpyHostToDevice) );

    cudaMalloc2E((void**)&R35_hst, R3, R5, R_size);
    //cudaCHK( hipMemcpyToSymbol(HIP_SYMBOL(R35_dev), &R35_hst, sizeof(float*)) );

    // output variables === M N Z ===
    cudaCHK( hipMalloc(&Zdat_hst, size_hst[3]) );
    cudaCHK( hipMalloc(&Zmax_hst, size_hst[3]) );
    //cudaCHK( hipMemcpyToSymbol(HIP_SYMBOL(Z_dat_dev), &Zdat_hst, sizeof(float*)) );
    cudaCHK( hipMemcpy(Zdat_hst, Z_f, size_hst[3], hipMemcpyHostToDevice) );
    cudaCHK( hipMemcpy(Zmax_hst, Z_f, size_hst[3], hipMemcpyHostToDevice) );

    cudaCHK( hipMalloc(&Zout_hst, size_hst[3]) );
    cudaCHK( hipMemcpyToSymbol(HIP_SYMBOL(Z_out_dev), &Zout_hst, sizeof(float*)) );


    cudaMalloc2E((void**)&MNdat_hst, NULL, NULL, size_hst[3]);
    //cudaCHK( hipMemcpyToSymbol(HIP_SYMBOL(MN_dat_dev), &MNdat_hst, sizeof(float*)) );

    cudaMalloc2E((void**)&MNout_hst, NULL, NULL, size_hst[3]);
    cudaCHK( hipMemcpyToSymbol(HIP_SYMBOL(MN_out_dev), &MNout_hst, sizeof(float*)) );

    // copy data into variables
    // const variables === H, size ===
    cudaCHK( hipMemcpy(H_hst, H_f, size_hst[3], hipMemcpyHostToDevice) );
    //cudaCHK( hipMemcpyToSymbol(HIP_SYMBOL(H_dev), &H_hst, sizeof(float*)) );

    cudaCHK( hipMemcpyToSymbol(HIP_SYMBOL(size_dev), &size_hst, sizeof(size_hst)) );

    //Texture Memories


    // kernel configurations
    DimGridMomt_MN   = dim3((size_hst[0]-1)/(31*(BLOCKX_MOMT>>5)) + 1, LOAD_PER_SM*(uint32_t)dev_prop.multiProcessorCount, 1);
    DimGridMomt      = dim3((size_hst[0]-1)/EXECY + 1, (size_hst[1]-1)/EXECX + 1, 1);
    DimGridMass      = dim3((size_hst[0]-1)/(31*(BLOCKX_MASS>>5)) + 1, LOAD_PER_SM*(uint32_t)dev_prop.multiProcessorCount, 1);
    DimGridOpenBD_LR = dim3((size_hst[0]-1)/(31*(BLOCKX_OPENBD>>5)) + 1, 1, 1);
    DimGridOpenBD_TB = dim3((size_hst[1]-1)/(31*(BLOCKX_OPENBD>>5)) + 1, 1, 1);
    GridMaxAmp       = (size_hst[2]-1)/MAXAMP_BLOCK + 1;

    //streams
    for (size_t i = 0; i < NUMSTREAM; i++) {
        cudaCHK( hipStreamCreate(EXECstream+i) );
    }

    #ifdef DEBUG
        // hipError_t err;
        //
        // printf("==== R2 ele chk===\n");
        // prt_mat(R2_f, size_hst[0], size_hst[1]);
        // printf("%e\n", R2_f[ID_hst(CHKR, CHKC)]);
        //
        // printf("==== R3 ele chk===\n" );
        // prt_mat(R3_f, size_hst[0], size_hst[1]);
        // printf("%e\n", R3_f[ID_hst(CHKR, CHKC)]);
        //
        // printf("==== H ele chk===\n");
        // prt_mat(H_f, size_hst[0], size_hst[1]);
        // printf("%e\n", H_f[ID_hst(CHKR, CHKC)]);
        //
        // printf("====R5===\n");
        // prt_mat(R5_f,size_hst[0], size_hst[1]);
        //
        // printf("====Z===\n" );
        // prt_mat(Z_f,size_hst[0], size_hst[1]);
        //
        // // printf("====R6===\n");
        // // for (size_t i = CHKC; i < CHKC+CHKSI; i++) {
        // //     printf("%e\t", R6_f[i]);
        // // }
        // // printf("\n");
        // //
        // // printf("====R1====\n");
        // // prt_mat(R1_f,size_hst[0], size_hst[1]);
        // //
        // // printf("====R11====\n");
        // // prt_mat(R11_f,size_hst[0],size_hst[1]);
        //
        //
        // GPUmemtest<<< 1, 1>>>();
        // hipDeviceSynchronize();
        // err = hipGetLastError();
        // cudaERROR(err);
        tmpout = (float*) malloc(2*size_hst[3]);
    #endif

    free(R_MASS);
    free(R3);
    free(R5);
}

extern "C" void cuda_update_(void) {
    float *tmp;
    // similar to function change
    cudaCHK( hipMemcpyToSymbol(HIP_SYMBOL(MN_out_dev), &MNdat_hst, sizeof(float*)) );
    //cudaCHK( hipMemcpyToSymbol(HIP_SYMBOL(MN_dat_dev), &MNout_hst, sizeof(float*)) );
    tmp = MNout_hst;
    MNout_hst = MNdat_hst;
    MNdat_hst = tmp;

    cudaCHK( hipMemcpyToSymbol(HIP_SYMBOL(Z_out_dev), &Zdat_hst, sizeof(float*)) );
    // cudaCHK( hipMemcpyToSymbol(HIP_SYMBOL(Z_dat_dev), &Zout_hst, sizeof(float*)) );
    tmp = Zout_hst;
    Zout_hst = Zdat_hst;
    Zdat_hst = tmp;
}

extern "C" void cuda_shutdown_(void){
    // Free Device Memory
    // Unbind Texture Memory
}


void cudaMalloc2E(void** cu_hst, void* e1, void* e2, size_t size){
    cudaCHK( hipMalloc(cu_hst, 2*size) );
    if (!e1) {
        cudaCHK( hipMemset(*cu_hst, 0, size) );
    }
    else{
        cudaCHK( hipMemcpy(*cu_hst, e1, size, hipMemcpyHostToDevice) );
    }
    if (!e2) {
        cudaCHK( hipMemset((void*)((char*)*cu_hst+size), 0, size) );
    }else{
        cudaCHK( hipMemcpy((void*)((char*)*cu_hst+size), e2, size, hipMemcpyHostToDevice) );
    }
}

#ifdef DEBUG
    __global__ void CHECK_VAR(){
        for (size_t i = 0; i < size_dev[0]; i++) {
            printf("%e\t",Z_out_dev[i]);
        }
        printf("\n" );
    }
    extern "C" void cmpvar_(const float *var_f, int *Case){
        if (*Case == 0) {
            printf("copying data\n");
            for (size_t i = 0; i < size_hst[2]; i++) {
                tmpout[i] = var_f[i];
            }
            return;
        }else{
            printf("comparing data\n" );
            for (size_t i = 0; i < size_hst[2]; i++) {
                if (abs(tmpout[i] - var_f[i]) > ERROR) {
                    printf("VAR[%d,%d] VAR_cu:%e VAR_f:%e %e\n", i%size_hst[0], i/size_hst[0] , tmpout[i], var_f[i], tmpout[i] - var_f[i]);
                }
            }
        }
    }
    // __global__ void GPUmemtest() {
    //     printf("%d %d %d %d\n",size_dev[0], size_dev[1], size_dev[2], size_dev[3]);
    //     printf("xxxxx GPU H ele chk xxxxx\n" );
    //     prt_mat(H_dev, size_dev[0], size_dev[1]);
    //     printf("%f\n", H_dev[ID(CHKR, CHKC)]);
    //
    //     printf("xxxxx GPU R2 ele chk xxxxx\n" );
    //     prt_mat(R24_dev, size_dev[0], size_dev[1]);
    //     printf("%f\n", R24_dev[ID(CHKR, CHKC)]);
    //
    //     printf("xxxxx GPU R3 ele chk xxxxx\n");
    //     for (size_t i = CHKC; i < CHKC+CHKSI; i++) {
    //         printf("%8.6e\t", R35_dev[i]);
    //     }
    // }


    __host__  __device__ void prt_mat(float *mat, size_t row, size_t col){
        printf("Matrix[%d:%d][%d:%d]\n", CHKR,CHKR+CHKSI, CHKC,CHKC+CHKSI);
        for (size_t i = CHKR; i < CHKR+CHKSI; i++) {
            for (size_t j = CHKC; j < CHKC+CHKSI; j++) {
                printf("%8.6e\t",  mat[j*row + i]);
            }
            printf("\n");
        }
    }
#endif
