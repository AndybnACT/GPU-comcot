#include "hip/hip_runtime.h"
#include "GPUHeader.h"
#include "GPUConfig.h"

// global variables-----------------------------
struct GPU_Layer Layer_struct[MAX_LAYERS];
uint32_t all_size[MAX_LAYERS][4];
__constant__ __device__ uint32_t all_size_dev[MAX_LAYERS][4];

hipStream_t EXECstream[NUMSTREAM];
dim3 DimBlockMomt_MN(BLOCKX_MOMT,1,1);
dim3 DimGridMomt_MN(0,1,1);
dim3 DimBlockMomt(BLOCKX,BLOCKY,2);
dim3 DimGridMomt(0,0,1);
dim3 DimBlockMass(BLOCKX_MASS,1,1);
dim3 DimGridMass(1,1,1);
dim3 DimBlockOpenBD(BLOCKX_OPENBD,1,1);
dim3 DimGridOpenBD_LR(0,1,1);
dim3 DimGridOpenBD_TB(0,1,1);
size_t GridMaxAmp;

hipDeviceProp_t dev_prop;
#ifdef DEBUG
    float *tmpout;
#endif
//----------------------------------------------
extern "C" void cuda_update_layer_(int *);
void cudaMalloc2E(void**, void*, void*, size_t);
#ifdef DEBUG
    __host__ __device__ void prt_mat(float*, size_t, size_t);
    __global__ void GPUmemtest(void);
    __global__ void CHECK_VAR(void);
    extern "C" void cmpvar_(const float *, int *);
#endif

extern "C" void cuda_alloc_layer(struct GPU_Layer *L, float *R1_f, float *R2_f, 
                                 float *R3_f, float *R4_f, float *R5_f, float *R6_f, 
                                 float *R11_f, float *H_f, float *Z_f, int *row, int *col){

    //float *R24_hst, *R35_hst, *H_hst;
    float *R3, *R5;
    float *R_MASS;

    L->l_size[0] = *row;
    L->l_size[1] = *col;
    L->l_size[2] = (*row)*(*col);
    L->l_size[3] = (*row)*(*col)*sizeof(float);
    // allocate variables
    // const === R2 R4 H ===
    cudaCHK( hipMalloc(&L->H_hst, L->l_size[3]) );

    cudaMalloc2E((void**)&L->R24_hst, R2_f, R4_f, L->l_size[3]);
    //cudaCHK( hipMemcpyToSymbol(HIP_SYMBOL(R24_dev), &R24_hst, sizeof(float*)) );
    // const variables R3 R5, R1 R6 R11
    size_t R_size = sizeof(float)*L->l_size[1];
    //R1 = (float*) malloc(R_size);
    R3 = (float*) malloc(R_size);
    R5 = (float*) malloc(R_size);
    //R11 =(float*) malloc(R_size);
    R_MASS = (float*) malloc(4*R_size);

    for (size_t i = 0; i < L->l_size[1]; i++) {
        R3[i] = R3_f[i*L->l_size[0]];
        R5[i] = R5_f[i*L->l_size[0]];
        R_MASS[4*i]   = R1_f[i*L->l_size[0]];
        R_MASS[4*i+1] = R6_f[i];
        R_MASS[4*i+2] = R11_f[i*L->l_size[0]];
    }

    cudaCHK( hipMalloc(&L->R_MASS_hst, 4*R_size) );
    cudaCHK( hipMemcpy(L->R_MASS_hst, R_MASS, 4*R_size, hipMemcpyHostToDevice) );

    cudaMalloc2E((void**)&L->R35_hst, R3, R5, R_size);

    // output variables === M N Z ===
    cudaCHK( hipMalloc(&L->Zdat_hst, L->l_size[3]) );
    cudaCHK( hipMalloc(&L->Zmax_hst, L->l_size[3]) );
    cudaCHK( hipMemcpy(L->Zdat_hst, Z_f, L->l_size[3], hipMemcpyHostToDevice) );
    cudaCHK( hipMemcpy(L->Zmax_hst, Z_f, L->l_size[3], hipMemcpyHostToDevice) );
    cudaCHK( hipMalloc(&L->Zout_hst, L->l_size[3]) );

    cudaMalloc2E((void**)&L->MNdat_hst, NULL, NULL, L->l_size[3]);
    cudaMalloc2E((void**)&L->MNout_hst, NULL, NULL, L->l_size[3]);

    // copy data into variables
    // const variables === H, size ===
    cudaCHK( hipMemcpy(L->H_hst, H_f, L->l_size[3], hipMemcpyHostToDevice) );

    int *ptr;
    memcpy(all_size[L->lid], L->l_size, 4*sizeof(uint32_t));
    cudaCHK( hipMalloc(&ptr, sizeof(all_size)) );
    cudaCHK( hipMemcpy(ptr, all_size, sizeof(all_size), hipMemcpyHostToDevice) );
    cudaCHK( hipMemcpyToSymbol(HIP_SYMBOL(all_size_dev), ptr, sizeof(all_size)) );
    cudaCHK( hipFree(ptr) );
    
    // kernel configurations
    L->DimGridMomt_MN   = dim3((L->l_size[0]-1)/(31*(BLOCKX_MOMT>>5)) + 1, LOAD_PER_SM*(uint32_t)dev_prop.multiProcessorCount, 1);
    L->DimGridMomt      = dim3((L->l_size[0]-1)/EXECY + 1, (L->l_size[1]-1)/EXECX + 1, 1);
    L->DimGridMass      = dim3((L->l_size[0]-1)/(31*(BLOCKX_MASS>>5)) + 1, LOAD_PER_SM*(uint32_t)dev_prop.multiProcessorCount, 1);
    L->DimGridOpenBD_LR = dim3((L->l_size[0]-1)/(31*(BLOCKX_OPENBD>>5)) + 1, 1, 1);
    L->DimGridOpenBD_TB = dim3((L->l_size[1]-1)/(31*(BLOCKX_OPENBD>>5)) + 1, 1, 1);
    L->GridMaxAmp       = (L->l_size[2]-1)/MAXAMP_BLOCK + 1;

    #ifdef DEBUG
        tmpout = (float*) malloc(2*l_size[3]);
    #endif

    free(R_MASS);
    free(R3);
    free(R5);
}

extern "C" void cuda_update_layer_(int *lid) {
    struct GPU_Layer *L = ldlayer(*lid);
    float *tmp;
    // similar to function change
    tmp = L->MNout_hst;
    L->MNout_hst = L->MNdat_hst;
    L->MNdat_hst = tmp;

    tmp = L->Zout_hst;
    L->Zout_hst = L->Zdat_hst;
    L->Zdat_hst = tmp;
}

extern "C" void gcomcot_init_gpu_(void)
{
    cudaCHK( hipGetDeviceProperties(&dev_prop, 0) );
    printf("GPU INFORMATIONS:                      %s\n", dev_prop.name);
    printf("-->Compute Capabilities [Major.Miner]: %d.%d\n", dev_prop.major, dev_prop.minor);
    printf("-->Clock Rate:                         %d\n", dev_prop.clockRate);
    printf("-->Streaming Multi-Processor Count:    %d\n", dev_prop.multiProcessorCount);
    printf("-->Shared Memory size per SM           %d\n", dev_prop.sharedMemPerBlock);
    printf("-->Total Constant Memory size:         %d\n", dev_prop.totalConstMem );
    printf("-->Maximum Grid Size:                  %dx%dx%d\n", dev_prop.maxGridSize[0],dev_prop.maxGridSize[1],dev_prop.maxGridSize[2]);
    printf("-->Warp Size:                          %d\n", dev_prop.warpSize);
    
    //streams
    for (size_t i = 0; i < NUMSTREAM; i++) {
        cudaCHK( hipStreamCreate(EXECstream+i) );
    }
}


extern "C" void gcomcot_init_layer_(int *layerid, int *parent, int *level,
                                
                                float *R1_f, float *R2_f,
                                float *R3_f, float *R4_f, float *R5_f, 
                                float *R6_f, float *R11_f, float *H_f, 
                                float *Z_f, int *row, int *col)
{
    struct GPU_Layer *L = ldlayer(*layerid);
    
    if (*layerid < 0) {
        printf("ERROR: invalid layerid\n");
        exit(-1);
    }
    if (*layerid >= MAX_LAYERS) {
        printf("ERROR: number of layer exceed MAX_LAYERS\n");
        exit(-1);
    }
    printf("Initializing layer id %d\n", *layerid);
    
    L->lid = *layerid;
    L->plid = *parent;
    L->lvl = *level;
    
    cuda_alloc_layer(L, R1_f, R2_f, R3_f, R4_f, R5_f,
                     R6_f, R11_f, H_f, Z_f, row, col);
    
    
}

extern "C" void cuda_shutdown_(void){
    // Free Device Memory
    // Unbind Texture Memory
}


void cudaMalloc2E(void** cu_hst, void* e1, void* e2, size_t size){
    cudaCHK( hipMalloc(cu_hst, 2*size) );
    if (!e1) {
        cudaCHK( hipMemset(*cu_hst, 0, size) );
    }
    else{
        cudaCHK( hipMemcpy(*cu_hst, e1, size, hipMemcpyHostToDevice) );
    }
    if (!e2) {
        cudaCHK( hipMemset((void*)((char*)*cu_hst+size), 0, size) );
    }else{
        cudaCHK( hipMemcpy((void*)((char*)*cu_hst+size), e2, size, hipMemcpyHostToDevice) );
    }
}

#ifdef DEBUG
__global__ void CHECK_VAR(){
    for (size_t i = 0; i < size_dev[0]; i++) {
        printf("%e\t",Z_out_dev[i]);
    }
    printf("\n" );
}
extern "C" void cmpvar_(const float *var_f, int *Case){
    if (*Case == 0) {
        printf("copying data\n");
        for (size_t i = 0; i < l_size[2]; i++) {
            tmpout[i] = var_f[i];
        }
        return;
    }else{
        printf("comparing data\n" );
        for (size_t i = 0; i < l_size[2]; i++) {
            if (abs(tmpout[i] - var_f[i]) > ERROR) {
                printf("VAR[%d,%d] VAR_cu:%e VAR_f:%e %e\n", i%l_size[0], i/l_size[0] , tmpout[i], var_f[i], tmpout[i] - var_f[i]);
            }
        }
    }
}


__host__  __device__ void prt_mat(float *mat, size_t row, size_t col){
    printf("Matrix[%d:%d][%d:%d]\n", CHKR,CHKR+CHKSI, CHKC,CHKC+CHKSI);
    for (size_t i = CHKR; i < CHKR+CHKSI; i++) {
        for (size_t j = CHKC; j < CHKC+CHKSI; j++) {
            printf("%8.6e\t",  mat[j*row + i]);
        }
        printf("\n");
    }
}
#endif /* DEBUG */
