#include "hip/hip_runtime.h"
#include "GPUHeader.h"
#include "GPUConfig.h"

#include "GPUOpen_BD.h"

extern "C" void openbd_launch_(float *Z_f_complete) {
    /* Only for outest layer, assume its layerid = 0 */
    struct GPU_Layer *L = ldlayer(1);

    hipError_t err;
    openbd_kernel<<< L->DimGridOpenBD_LR, DimBlockOpenBD, 0, EXECstream[0] >>>(*L, LEFT);// MN has been changed (:,:,1) <--> (:,:,2)
    openbd_kernel<<< L->DimGridOpenBD_LR, DimBlockOpenBD, 0, EXECstream[1] >>>(*L, RIGHT);// so use M(:,:,1) directly
    openbd_kernel<<< L->DimGridOpenBD_TB, DimBlockOpenBD, 0, EXECstream[2] >>>(*L, TOP);
    openbd_kernel<<< L->DimGridOpenBD_TB, DimBlockOpenBD, 0, EXECstream[3] >>>(*L, BOTTOM);
    hipDeviceSynchronize();
    err = hipGetLastError();
    cudaERROR(err);


    #ifdef DEBUG
        printf("printing information for debugging\n" );
        cudaCHK( hipMemcpy(tmpout, Zout_hst, size_hst[3], hipMemcpyDeviceToHost) );
        for (size_t i = 0; i < size_hst[2]; i++) {
            if (abs(tmpout[i] - Z_f_complete[i]) > ERROR) {
                printf("Z[%d,%d] Z_cu:%e Z_f:%e %e\n", i%size_hst[0], i/size_hst[0] , tmpout[i], Z_f_complete[i], tmpout[i] - Z_f_complete[i]);
            }
        }
    #endif

}


__global__ void openbd_kernel(struct GPU_Layer L, bdside BOUNDARY) {
    
    const float* __restrict__ MN = L.MNdat_hst;
    const float* __restrict__ H = L.H_hst;
    float* __restrict__ Z_out_dev = L.Zout_hst;
    const uint32_t __restrict__ *size_dev = all_size_dev[L.lid];
    
    #define UB 99.0
    float ztmp=0.0, h, m, n, cc;
    uint32_t row=0, col=0;
    switch (BOUNDARY) {
        case RIGHT:
            col = (size_dev[1]-1)*size_dev[0];
            row = blockIdx.x*31*(blockDim.x>>5) + 31*(threadIdx.x>>5) + threadIdx.x%32;
            if (row < size_dev[0]-1) {
                h =  H[row+col];
                m = MN[row+col];// must not load in the following if block, or 1st lane would get 0
                float m_suf = __shfl_up_sync(0xFFFFFFFF,m,1);// must not shuffle in the following if block, or 1st lane would get 0
                n = MN[row+col-size_dev[0]+size_dev[2]];
                cc = 1/sqrtf(GRAV*h);
                if (threadIdx.x % 32 != 0) {
                    if (h > GX) {
                        float uh_2 = 0.25*(m+m_suf)*(m+m_suf);
                        ztmp = sqrtf(n*n + uh_2)*cc;
                        if (n < 0.0) ztmp *= -1;
                        if (ztmp > UB || ztmp < -UB) ztmp = 0.0;
                    }// else {ztmp=0.0;}
                    Z_out_dev[row+col] = ztmp;
                }
                else if (row == 0) { // --|
                    if (h > GX) {
                        ztmp = sqrtf(m*m + n*n)*cc;
                        if (m > 0 || n < 0) ztmp *= -1;
                        if (ztmp > UB || ztmp < -UB) ztmp = 0.0;
                    }
                    Z_out_dev[col] = ztmp;
                }
            }
            break;
        case LEFT:
            row = blockIdx.x*31*(blockDim.x>>5) + 31*(threadIdx.x>>5) + threadIdx.x%32;
            if (row < size_dev[0]-1) {
                h =  H[row];
                m = MN[row];
                float m_suf = __shfl_up_sync(0xFFFFFFFF,m,1);
                n = MN[row+size_dev[2]];
                cc = 1/sqrtf(GRAV*h);
                if (threadIdx.x % 32 != 0) {
                    if (h > GX) {
                        float uh_2 = 0.25*(m+m_suf)*(m+m_suf);
                        ztmp = sqrtf(n*n + uh_2)*cc;
                        if (n > 0.0) ztmp *= -1;
                        if (ztmp > UB || ztmp < -UB) ztmp = 0.0;
                    }// else {ztmp=0.0;}
                    Z_out_dev[row] = ztmp;
                }
                else if (row == 0) {//  |--
                    if (h > GX) {
                        ztmp = sqrtf(m*m + n*n)*cc;
                        if (m > 0 || n < 0) ztmp *= -1;
                        if (ztmp > UB || ztmp < -UB) ztmp = 0.0;
                    }
                    Z_out_dev[col] = ztmp;
                }
            }
            break;
        case TOP: // should use texture
            col = blockIdx.x*31*(blockDim.x>>5) + 31*(threadIdx.x>>5) + threadIdx.x%32;
            if (col < size_dev[1]-1 && col > 0) {
                col *= size_dev[0];
                h =  H[col];
                n = MN[col+size_dev[2]];
                float n_suf = __shfl_up_sync(0xFFFFFFFF,n,1);
                if (threadIdx.x%32 != 0) {
                    if (h > GX) {
                        m = MN[col];
                        cc = 1/sqrtf(GRAV*h);
                        float uh_2 = 0.25*(n+n_suf)*(n+n_suf);

                        ztmp = sqrtf(uh_2 + m*m)*cc;
                        if (m > 0.0) ztmp *= -1;
                        if (ztmp > UB || ztmp < -UB) ztmp = 0.0;
                    }
                    Z_out_dev[col] = ztmp;
                }
            }
            break;
        case BOTTOM:
            row = size_dev[0]-1;
            col = blockIdx.x*31*(blockDim.x>>5) + 31*(threadIdx.x>>5) + threadIdx.x%32;

            if (col < size_dev[1]-1 && col > 0) { //bottom body
                col *= size_dev[0];
                h =  H[col+row];
                n = MN[col+row+size_dev[2]];
                float n_suf = __shfl_up_sync(0xFFFFFFFF,n,1);
                if (threadIdx.x%32 != 0) {
                    if (h > GX) {
                        m = MN[col+row-1];
                        cc = 1/sqrtf(GRAV*h);
                        float uh_2 = 0.25*(n+n_suf)*(n+n_suf);

                        ztmp = sqrtf(uh_2 + m*m)*cc;
                        if (m < 0.0) ztmp *= -1;
                        if (ztmp > UB || ztmp < -UB) ztmp = 0.0;
                    }
                    Z_out_dev[col+row] = ztmp;
                }
            }else if (col == 0 || col == size_dev[1]-1){ // bottom boundary
                uint32_t id = row+col*size_dev[0];
                h =  H[id];
                if (h > GX) {
                    cc = 1/sqrtf(GRAV*h);
                    m = MN[id-1];
                    if (col == 0) {  // |__
                        n = MN[row+size_dev[2]];
                        ztmp = sqrtf(m*m + n*n);
                        ztmp *= cc;
                        if (m < 0.0 || n > 0.0) ztmp *= -1;
                        if (ztmp > UB || ztmp < -UB) ztmp = 0.0;
                    }else{           //  __|
                        n = MN[row + (col-1)*size_dev[0] + size_dev[2]];
                        ztmp = sqrtf(m*m + n*n);
                        ztmp *= cc;
                        if (m < 0.0 || n < 0.0) ztmp *= -1;
                        if (ztmp > UB || ztmp < -UB) ztmp = 0.0;
                    }
                }
                Z_out_dev[id] = ztmp;
            }
    }
}
